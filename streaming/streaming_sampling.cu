#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <iterator>
#include <queue>
#include <random>
#include <set>
#include "gpu_graph.cuh"
#include "graph.h"
#include "herror.h"
#include "sampler.cuh"
#include "wtime.h"
using namespace std;

// int RAND_MAX=10000;
int sum(int length, int *a) {
  int total = 0;
  // std::cout<<"\n size:"<<length<<"\n";
  for (int i = 0; i < length; i++) {
    // std::cout<<a[i]<<"\n";
    total += a[i];
  }
  // std::cout<<"Total:"<< total <<"\n";
  return total;
}

void display(int *a, int length) {
  for (int i = 0; i < length; i++) {
    printf("%d\n", a[i]);
  }
}

__device__ void d_display(int *a, int *b, int length) {
  for (int i = 0; i < length; i++) {
    printf("Node:%d, Edge:%d\n", a[i], b[i]);
  }
}

void prefix_sum(int length, int *prefix_list) {
  for (int i = 1; i < length; i++) {
    prefix_list[i] = (prefix_list[i - 1] + prefix_list[i]);
    // printf("%dth sum, %f +  %d =
    // %f\n",i,prefix_list[i-1],arr[i],prefix_list[i]);
  }
}

__device__ int binary_search(int start, int end, float value, float *arr) {
  // printf("low:%d,high:%d,value:%f\n",start,end,value);
  int low = start;
  int high = end;
  int index = start;
  while (low <= high) {
    index = ((low + high) / 2);
    if (value < arr[index]) {
      // set high to index-1
      high = index - 1;
      // printf("high:%d\n",high);
    } else if (value > arr[index]) {
      // set low to index+1
      low = index + 1;
      // printf("low:%d\n",low);

    } else {
      break;
    }
  }
  return index;
}

__device__ int bitmap_binary_search(int start, int end, float value, float *arr,
                                    int *bitmap, int bitmap_start, int &is_in) {
  // printf("low:%d,high:%d,value:%f\n",start,end,value);
  int low = start;
  int high = end;
  int index = start;
  int bitmap_width = 32;
  while (low <= high) {
    index = ((low + high) / 2);
    if (value < arr[index]) {
      // set high to index-1
      high = index - 1;
      // printf("high:%d\n",high);
    } else if (value > arr[index]) {
      // set low to index+1
      low = index + 1;
      // printf("low:%d\n",low);
    } else {
      break;
    }
  }
  int bitmap_pos = index;
  int bit_block_index =
      bitmap_pos / bitmap_width;                  // find the address of bitmap
  int bit_block_pos = bitmap_pos % bitmap_width;  // position within a address
  // reversed------------

  // int bit_block_pos = bitmap_pos / bitmap_width;
  // int bit_block_index= bitmap_pos % bitmap_width;
  int initial_mask = 1;
  int mask = (initial_mask << bit_block_pos);
  int status = atomicOr(&bitmap[bit_block_index + bitmap_start], mask);
  is_in = (mask & status) >> bit_block_pos;

  // is_in= 0x00000001 & (status >> bit_block_pos);
  // printf("thread: %d, index:%d, bit_block_index:%d, bit_block_pos:%d,
  // mask:%d, status: %d,shift: %d,
  // is_in:%d\n",threadIdx.x,index,bit_block_index,bit_block_pos,mask,status,(mask
  // & status),is_in);
  return index;
}

void r2() {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0, 1);
  for (int n = 0; n < 10; ++n) {
    std::cout << dis(gen) << ' ';
  }
}

__device__ float frandom(hiprandState *global) {
  // hiprand_init(1000,threadIdx.x,10,&global[threadIdx.x]);
  float x = ((hiprand_uniform(&global[0])));
  return x;
}

__device__ int linear_search(int neighbor, int *partition1, int *bin_count,
                             int bin, int BIN_OFFSET, int BIN_START,
                             int BUCKETS) {
  int len = bin_count[bin + BIN_OFFSET];

  int i = bin + BIN_START;
  // printf("\nL: %d, I:%d\n",len,i);
  int step = 0;
  while (step < len) {
    int test = partition1[i];
    // printf("Neighbor: %d, Test: %d, address: %d\n",neighbor,test,i);
    if (test == neighbor) {
      // printf("Duplicate detected
      // -------------------------------------------------------\n");
      return 1;
    } else {
      i += BUCKETS;
    }
    step += 1;
  }
  return 0;
}

__device__ void gpu_prefix(int total_step, int warp_tid, float *degree_l,
                           int offset_d_n, int warpsize, int len) {
  for (int i = 0; i < total_step; i++) {
    // Loop the threads
    int req_thread = len / (powf(2, (i + 1)));
    for (int iid = warp_tid; iid <= req_thread; iid += warpsize) {
      int tid_offset = iid * powf(2, i + 1);
      // calculate the index
      int i1 = (tid_offset) + (powf(2, i)) - 1 + offset_d_n;
      int i2 = (tid_offset) + powf(2, i + 1) - 1 + offset_d_n;
      if (i1 > (offset_d_n + len - 1)) {
        break;
      }
      // printf("i:%d, Index1 %d: %f,Index2 %d: %f,
      // thread:%d\n",i,i1,degree_l[i1],i2,degree_l[i2],threadIdx.x);
      // load the values to shared mem
      int temp1 = degree_l[i1];
      int temp2 = degree_l[i2];
      degree_l[i2] = temp2 + temp1;
      // printf("Index:%d, Value:%d \n",i2,temp[i2]);
    }
  }
  degree_l[len - 1 + offset_d_n] = 0;
  // printf("\nDownstep:%d\n",degree_l[len-1]);
  for (int i = (total_step - 1); i >= 0; i--) {
    // Loop the threads
    int req_thread = len / (powf(2, (i + 1)));
    for (int iid = warp_tid; iid <= req_thread; iid += warpsize) {
      int tid_offset = iid * powf(2, i + 1);
      int i1 = (tid_offset) + (powf(2, i)) - 1 + offset_d_n;
      int i2 = (tid_offset) + powf(2, i + 1) - 1 + offset_d_n;
      if (i1 > (offset_d_n + len - 1)) {
        break;
      }
      //  printf("temp1: %d, temp2: %d, thread:%d\n",i1,i2,threadIdx.x);
      // printf("Index1 %d: %f,Index2 %d: %f,
      // thread:%d\n",i1,degree_l[i1],i2,degree_l[i2],threadIdx.x);
      int temp1 = degree_l[i1];
      int temp2 = degree_l[i2];
      degree_l[i1] = temp2;
      degree_l[i2] = temp2 + temp1;
      // printf("Index:%d, Value:%d \n",i2,temp[i2]);
    }
  }
}

__global__ void check(int Graph_block_size, int streamid, int block_id,
                      vertex_t *adj_list, index_t *beg_pos,
                      weight_t *weight_list, int vertex_count,
                      hiprandState *global_state, int *g_node_list,
                      int *g_edge_list, int *neigh_l, float *degree_l,
                      int n_blocks, int *d_seed, int n_threads, int *total,
                      int *hashtable, int *bitmap, int total_subgraphs,
                      int *node, int *queue, int *sample_id, int *depth_tracker,
                      int *qstart_global, int *qstop_global, int *g_sub_index,
                      int n_child, int depth_limit, int sample_size, int queue_size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // int __shared__ q1_start, q2_end, depth, q2_start, q2_stop;
  int temp_queue_start = qstart_global[block_id];
  int temp_queue_stop = qstop_global[block_id];
  //-----------------We may require a barrier here for storing temp
  //queue---------------------//
  int __shared__ bin_count[128];
  int warp_tid = threadIdx.x % 32;
  int G_warpID = tid / 32;
  int warpId = threadIdx.x / 32;
  int warpsize = 32;
  int offset_d_n = G_warpID * 4000;
  int BUCKETS = 32;
  int BINsize = BUCKETS * 6;
  int bitmap_size = 100;
  int Graph_block = 4;
  float prefix_time, local_d_time, global_d_time;
  clock_t start_time, stop_time;
  int __shared__ prefix;
  int seed_index;
  int BIN_OFFSET = 0;
  int depthcount, edges_traversed, q_stop, vertex, total_work;
  int q_start;
  int queue_start_address = block_id * queue_size;
  hiprandState local_state = global_state[threadIdx.x];
  hiprand_init(
      tid, 0, 0,
      &local_state);  // sequence created with different seed and same sequence
  int depth_flag = 0;
  edges_traversed = 0;
  // add all items to the combined queue: Number of threads must be greater than
  // samples
  if ((qstop_global[block_id] - qstart_global[block_id]) != 0) {
    if (warp_tid == 0) {
      q_start = atomicAdd(&qstart_global[block_id], 1);
    }
    q_start = __shfl_sync(0xffffffff, q_start, 0);
    __syncwarp();

    while (q_start < qstop_global[block_id]) {
      vertex = queue[q_start + queue_start_address];
      //if(warp_tid==0){printf("Block_id:%d, StreamId: %d, G_warpID: %d,SampleID:%d, vertex:%d, q_stop:%d,q_start:%d,depth:%d\n",block_id,streamid,G_warpID,sample_id[q_start+queue_start_address],vertex,qstop_global[block_id],q_start,depth_tracker[q_start+queue_start_address]);}
      int neighbor_start = beg_pos[vertex];
      int neighbor_end = beg_pos[vertex + 1];
      int neighbor_length = neighbor_end - neighbor_start;
      edges_traversed += neighbor_length;
      if (neighbor_length == 0) {
        if (warp_tid == 0) {
          q_start = atomicAdd(&qstart_global[block_id], 1);
        }
        q_start = __shfl_sync(0xffffffff, q_start, 0);
        __syncwarp();
        continue;
      }
      int is_in = 0;
      int new_neighbor;
      int selected = 0;
      if (neighbor_length < n_child) {
        prefix = 0;
      } else {
        prefix = 1;
      }
      int thread_flag = 0;
      if ((warp_tid < n_child) && (warp_tid < neighbor_length)) {
        thread_flag = 1;
      }
      if (prefix) {
        // For each neighbor, calculate the degree of its neighbor
        int index = offset_d_n + warp_tid;  // use block and thread Id for index
        for (int i = warp_tid + neighbor_start; i < neighbor_end;
             i += warpsize) {
          // neighbor ID
          int temp = adj_list[i];
          // if((temp>Graph_block_size)& (warp_tid==0)){printf("Reading from
          // outside.\n");} degree of neighbor
          degree_l[index] = float(beg_pos[temp + 1] - beg_pos[temp]);
          // printf("%d has a degree of %f found by
          // %d,index:%d\n",temp,degree_l[index],threadIdx.x,index);
          index += warpsize;
        }
        int i_start_neigh = offset_d_n;
        int i_end_neigh = i_start_neigh + neighbor_length;
        //	printf("Starting prefix_sum\n");
        // start_time = clock();
        float bits = log2f(neighbor_length);
        int raise = ceilf(bits);
        int max_bit = powf(2, raise);
        int len = max_bit;
        int total_step = log2f(max_bit);
        gpu_prefix(total_step, warp_tid, degree_l, offset_d_n, warpsize, len);
        float sum = degree_l[neighbor_length - 1 + offset_d_n];
        for (int i = warp_tid + i_start_neigh; i < i_end_neigh; i += warpsize) {
          // printf("i:%d, degree:%.2f\n",i,degree_l[i]);
          degree_l[i] = degree_l[i] / ((double)sum);
        }
        // start_time = clock();
        int bitmap_start = G_warpID * bitmap_size;
        if (warp_tid < n_child) {
          float r = hiprand_uniform(&local_state);
          //------------------------------------Using
          //bitmaps----------------------------------------------
          selected =
              bitmap_binary_search(i_start_neigh, i_end_neigh, r, degree_l,
                                   bitmap, bitmap_start, is_in);
          new_neighbor = adj_list[selected + neighbor_start - offset_d_n];
          // if(is_in==0) {printf("Index: %d, New N: %d, Thread:
          // %d\n",selected,new_neighbor,threadIdx.x);}
          //--------------------------------------------------------------------------------------------
        }
        // Reset Bitmaps
        int start = bitmap_start + warp_tid;
        int end = bitmap_start + bitmap_size;
        for (int i = start; i < end; i += warpsize) {
          bitmap[i] = 0;
          // printf("Bitmap cleared at %d\n",i);
        }
      }
      else {
        if (thread_flag) {
          new_neighbor =
              adj_list[warp_tid + neighbor_start];  // unwanted thread also may
                                                    // get some child but will be
                                                    // neglected in next section
        }
        // printf("New Neighbor: %d, thread: %d\n",new_neighbor,threadIdx.x);
      }
      /* Use hashtable for detecting duplicates*/
      int BIN_START = sample_id[q_start] * BINsize;
      if (is_in == 0 && thread_flag) {
        int bin = new_neighbor % BUCKETS;
        is_in = linear_search(new_neighbor, hashtable, bin_count, bin,
                              BIN_OFFSET, BIN_START, BUCKETS);
        // if(is_in==1){printf("Duplicated Found: %d\n",new_neighbor);}
      }
      //-------------------------------------------------------------------
      if (is_in == 0 && thread_flag) {
        //------------------------Store in
        //hashtable-----------------------------//
        int bin = new_neighbor % BUCKETS;
        // int index= warpId;
        int index = atomicAdd(&bin_count[bin + BIN_OFFSET], 1);
        hashtable[index] = new_neighbor;
        hashtable[index * BUCKETS + bin + BIN_START] = new_neighbor;
        int g_sub_start = sample_id[q_start] * sample_size;
        int g_to = atomicAdd(&g_sub_index[sample_id[q_start]], 1);
        //g_node_list[g_to + g_sub_start] = vertex;
        //g_edge_list[g_to + g_sub_start] = new_neighbor;
        printf("%d,%d,%d,%d\n",vertex,new_neighbor,sample_id[q_start],depth_tracker[q_start + queue_start_address]);
	//Added to sample:752601,328138,20,0,2
	// add to the expand queue
        if (depth_tracker[q_start] < depth_limit) {
          int new_bin = new_neighbor / Graph_block_size;
          int new_queue_start = new_bin * queue_size;
	  // if(new_bin!=0)
          // { printf("Block:%d, Added to block:%d\n",block_id,new_bin);}
          int to = atomicAdd(&qstop_global[new_bin], 1);
          queue[to + new_queue_start] = new_neighbor;
          sample_id[to + new_queue_start] =
              sample_id[q_start + queue_start_address];
          depth_tracker[to + new_queue_start] =
              depth_tracker[q_start + queue_start_address] + 1;
         //printf("Added: %d,  to queue at index %d and block %d, local_index: %d, offset: %d, new_d: %d, prev_d: %d\n",new_neighbor,to + new_queue_start,new_bin, to, new_queue_start,depth_tracker[to + new_queue_start], depth_tracker[q_start + queue_start_address]);
	 }
      }
      // q_start+=1;
      if ((qstart_global[block_id] > qstop_global[block_id])) {
        break;
      }
      if (warp_tid == 0) {
        q_start = atomicAdd(&qstart_global[block_id], 1);
      }
      q_start = __shfl_sync(0xffffffff, q_start, 0);
      __syncwarp();
    }
  }
}

int build_histogram(int n_subgraph, int *input, int *frequency,
                    int block_window_size, int block_size, int vert_count,
                    int vertex_block_count) {
  int max_index = 0, max_value = 0;
  for (int i = 0; i < n_subgraph; i++) {
    int block = input[i] / block_size;
    if (block > vertex_block_count) {
      block = vertex_block_count;
    }
    // cout<<"Value:"<<input[i]<<"\tBlock:"<<block<<"\n";
    frequency[block] += 1;
  }
  // display(frequency,vertex_block_count);
  prefix_sum(vertex_block_count, frequency);
  // display(frequency,vertex_block_count);
  for (int j = 0; j < (vertex_block_count - 5); j += block_window_size) {
    int combined_freq = frequency[j + block_window_size - 1] - frequency[j];
    if (combined_freq > max_index) {
      max_index = j;
      max_value = combined_freq;
    }
  }
  cout << "Max_index:" << max_index << "Max_value:" << max_value << "\n";
  return max_index;
}

int block_augument(int blocks, int vertex_count, index_t *beg_pos,
                   int *beg_size_list, int *adj_size_list) {
  int block_size = (vertex_count) / blocks;
  for (int i = 0; i < (blocks + 1); i += 1) {
    int start_block = i * block_size;
    if (i == blocks) {
      start_block = vertex_count;
    }
    beg_size_list[i] = start_block;
    int start_adj = beg_pos[block_size * i];
    adj_size_list[i] = start_adj;
  }
  return 0;
}

struct arguments Sampler(char beg[100], char csr[100], int n_blocks,
                         int n_threads, int n_subgraph, int frontier_size,
                         int neighbor_size, int depth, struct arguments args,
                         int rank) {
  // if(args!=7){std::cout<<"Wrong input\n"; return -1;}
  //n_child, depth, each_subgraph, queue_size
  // cout<<"\nblocks:"<<n_blocks<<"\tThreads:"<<n_threads<<"\tSubgraphs:"<<n_subgraph<<"\n";
  // int n_threads=32;
  int *total = (int *)malloc(sizeof(int) * n_subgraph);
  int len = 5;
  int T_Group = n_threads / 32;
  int n_child = neighbor_size;
  int each_subgraph = depth * n_child;
  int total_length = each_subgraph * n_subgraph;
  int neighbor_length_max = n_blocks * 6000 * T_Group;
  int PER_BLOCK_WARP = T_Group;
  int BUCKET_SIZE = 10;
  int BUCKETS = 128;
  int total_mem_for_hash = n_blocks * PER_BLOCK_WARP * BUCKETS * BUCKET_SIZE;
  int total_mem_for_bitmap = n_blocks * PER_BLOCK_WARP * 300;
  int queue_size = 10000;
  int Graph_block = 4;
  int total_queue_memory = queue_size * Graph_block * n_subgraph;

  // std::cout<<"Input: ./exe beg csr nblocks nthreads\n";
  const char *beg_file = beg;
  const char *csr_file = csr;
  const char *weight_file = csr;  // unnecessary
  // template <file_vertex_t, file_index_t, file_weight_t
  // new_vertex_t, new_index_t, new_weight_t>
  graph<long, long, long, vertex_t, index_t, weight_t> *ginst =
      new graph<long, long, long, vertex_t, index_t, weight_t>(
          beg_file, csr_file, weight_file);
  int vertex_count = ginst->vert_count;
  int edge_count = ginst->edge_count;
  int Graph_block_size = vertex_count / Graph_block;
  // int Graph_block_size=2000;
  /*
  printf("Size of blocks\n");
  for (int i = 0; i < 4; i++) {
    printf("%d,%d\n", i,
           ginst->beg_pos[(i + 1) * Graph_block_size] -
               ginst->beg_pos[(i)*Graph_block_size]);
  }
  */
  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  gpu_graph ggraph(ginst);
  int *node_list = (int *)malloc(sizeof(int) * total_length);
  int *set_list = (int *)malloc(sizeof(int) * total_length);
  float *n_random = (float *)malloc(sizeof(float) * n_threads);
  int *seeds = (int *)malloc(sizeof(int) * total_queue_memory);
  int *seeds_counter = (int *)malloc(sizeof(int) * Graph_block);
  int *start_queue = (int *)malloc(sizeof(int) * Graph_block);
  int *degree_list = (int *)malloc(sizeof(int) * ginst->edge_count);
  int *adj_size_list = (int *)malloc(sizeof(int) * (Graph_block + 1));
  int *beg_size_list = (int *)malloc(sizeof(int) * (Graph_block + 1));
  for (int n = 0; n < Graph_block; n++) {
    seeds_counter[n] = 0;
    start_queue[n] = 0;
  }
  std::random_device rd;
  // 200 --> 370 Mteps
  int numBlocks;
  // hipGetDevice(&device);
  // hipGetDeviceProperties(&prop, device);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, check, n_threads,
                                                0);

  int deviceCount;
  HRR(hipGetDeviceCount(&deviceCount));
  printf("My rank: %d, totaldevice: %d\n", rank,deviceCount);
  HRR(hipSetDevice(rank%deviceCount));
  // cout<<"Max allocatable Blocks:"<<numBlocks<<"\n";
  int *d_node_list;
  int *d_edge_list;
  int *d_neigh_l;
  float *d_degree_l;
  // float *d_random;
  int *d_seed;
  int *d_total;
  for (int i = 0; i < (ginst->edge_count); i++) {
    int neighbor = ginst->adj_list[i];
    degree_list[i] = ginst->beg_pos[neighbor + 1] - ginst->beg_pos[neighbor];
  }
  int *hashtable, *bitmap, *node, *queue, *qstop_global, *qstart_global,
      *sample_id, *depth_tracker, *g_sub_index, *degree_l, *prefix_status;
  // Size of blocks
  HRR(hipMalloc((void **)&d_total, sizeof(int) * n_subgraph));
  HRR(hipMalloc((void **)&node, sizeof(int) * 2));
  HRR(hipMalloc((void **)&degree_l, sizeof(int) * ginst->edge_count));
  HRR(hipMalloc((void **)&prefix_status, sizeof(int) * ginst->edge_count));
  HRR(hipMalloc((void **)&d_degree_l, sizeof(float) * ginst->edge_count));
  HRR(hipMalloc((void **)&qstart_global, sizeof(int) * Graph_block));
  HRR(hipMalloc((void **)&qstop_global, sizeof(int) * Graph_block));
  HRR(hipMalloc((void **)&d_node_list, sizeof(int) * total_length));
  HRR(hipMalloc((void **)&d_edge_list, sizeof(int) * total_length));
  HRR(hipMalloc((void **)&d_neigh_l, sizeof(int) * neighbor_length_max));
  HRR(hipMalloc((void **)&hashtable, sizeof(int) * total_mem_for_hash));
  HRR(hipMalloc((void **)&bitmap, sizeof(int) * total_mem_for_bitmap));
  HRR(hipMalloc((void **)&d_degree_l, sizeof(float) * neighbor_length_max));
  HRR(hipMalloc((void **)&queue, sizeof(int) * total_queue_memory));
  HRR(hipMalloc((void **)&sample_id, sizeof(int) * total_queue_memory));
  HRR(hipMalloc((void **)&depth_tracker, sizeof(int) * total_queue_memory));
  HRR(hipMalloc((void **)&g_sub_index, sizeof(float) * total_queue_memory));
  int *h_sample_id = (int *)malloc(sizeof(int) * total_queue_memory);
  int *h_depth_tracker = (int *)malloc(sizeof(int) * total_queue_memory);
  std::mt19937 gen(57);
  std::uniform_int_distribution<> dis(1, vertex_count / 4);

  for (int n = 0; n < n_subgraph; n++) {
    int new_seed = dis(gen);
    int bin_new = new_seed / Graph_block_size;
    if (bin_new > Graph_block) {
      bin_new = Graph_block;
    }
    int pos = bin_new * (queue_size) + seeds_counter[bin_new];
    assert(pos < total_queue_memory);
    seeds_counter[bin_new]++;
    seeds[pos] = new_seed;
    h_sample_id[pos] = n;
    h_depth_tracker[pos] = 0;
    // printf("N_subgraph: %d, Seed:%d, Bin:%d\n",n,new_seed,bin_new);
  }
  /*	For streaming partition */

  HRR(hipMemcpy(queue, seeds, sizeof(int) * total_queue_memory,
                 hipMemcpyHostToDevice));
  HRR(hipMemcpy(qstart_global, start_queue, sizeof(int) * Graph_block,
                 hipMemcpyHostToDevice));
  HRR(hipMemcpy(qstop_global, seeds_counter, sizeof(int) * Graph_block,
                 hipMemcpyHostToDevice));
  HRR(hipMemcpy(sample_id, h_sample_id, sizeof(int) * total_queue_memory,
                 hipMemcpyHostToDevice));
  HRR(hipMemcpy(depth_tracker, h_depth_tracker,
                 sizeof(int) * total_queue_memory, hipMemcpyHostToDevice));
  // create three cuda streams

  hipStream_t stream1, stream2, stream3, stream4;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);
  hipEvent_t event;
  hipEventCreate(&event);
  // find top 3 blocks
  int sampling_complete = false;
  int i = 0, block_id1 = 0, block_id2 = 1, block_id3 = 2, block_id4 = 3;

  int q_count, max, value;
  block_augument(Graph_block, vertex_count, ginst->beg_pos, beg_size_list,
                 adj_size_list);
  int *block_active = (int *)malloc(sizeof(int) * (Graph_block));
  int *frontiers_count = (int *)malloc(sizeof(int) * (Graph_block));

  for (int j = 0; j < Graph_block; j++) {
    frontiers_count[j] = seeds_counter[j] - start_queue[j];
    // printf("Value: %d, j: %d,Q_count:\n",frontiers_count[j],j);
    if (frontiers_count[j] == 0) {
      block_active[j] = 0;
    } else {
      block_active[j] = 1;
    }
  }
  printf("<Sampled edges in csv format.>\nsource, destination, sample_id, depth\n");
  // display(block_active,Graph_block);
  // block[1]=1;
  // block[2]=1;
  // printf("Start while loop.\n");
  double time_start = wtime();
  while (sampling_complete == false) {
    // display(block_active,Graph_block);
    if (1) {
      H_ERR(hipMemcpyAsync(&ggraph.adj_list[adj_size_list[block_id1]],
                            &ginst->adj_list[adj_size_list[block_id1]],
                            adj_size_list[block_id2] - adj_size_list[block_id1],
                            hipMemcpyHostToDevice, stream1));
      H_ERR(hipMemcpyAsync(&ggraph.beg_pos[beg_size_list[block_id1]],
                            &ginst->beg_pos[beg_size_list[block_id1]],
                            beg_size_list[block_id2] - beg_size_list[block_id1],
                            hipMemcpyHostToDevice, stream1));

      check<<<n_blocks, n_threads, 0, stream1>>>(
          Graph_block_size, 0, block_id1, ggraph.adj_list, ggraph.beg_pos,
          ggraph.weight_list, ggraph.vert_count, d_state, d_node_list,
          d_edge_list, d_neigh_l, d_degree_l, n_blocks, d_seed, n_threads,
          d_total, hashtable, bitmap, n_subgraph, node, queue, sample_id,
          depth_tracker, qstart_global, qstop_global, g_sub_index, 
	  n_child, depth, each_subgraph, queue_size);
    }

    if (block_active[1]) {
      H_ERR(hipMemcpyAsync(&ggraph.adj_list[adj_size_list[block_id2]],
                            &ginst->adj_list[adj_size_list[block_id2]],
                            adj_size_list[block_id3] - adj_size_list[block_id2],
                            hipMemcpyHostToDevice, stream2));
      H_ERR(hipMemcpyAsync(&ggraph.beg_pos[beg_size_list[block_id2]],
                            &ginst->beg_pos[beg_size_list[block_id2]],
                            beg_size_list[block_id3] - beg_size_list[block_id2],
                            hipMemcpyHostToDevice, stream2));
      check<<<n_blocks, n_threads, 1, stream2>>>(
          Graph_block_size, 1, block_id2, ggraph.adj_list, ggraph.beg_pos,
          ggraph.weight_list, ggraph.vert_count, d_state, d_node_list,
          d_edge_list, d_neigh_l, d_degree_l, n_blocks, d_seed, n_threads,
          d_total, hashtable, bitmap, n_subgraph, node, queue, sample_id,
          depth_tracker, qstart_global, qstop_global, g_sub_index,
	  n_child, depth, each_subgraph, queue_size);
    }

    if (block_active[2]) {
      H_ERR(hipMemcpyAsync(&ggraph.adj_list[adj_size_list[block_id3]],
                            &ginst->adj_list[adj_size_list[block_id3]],
                            adj_size_list[block_id4] - adj_size_list[block_id3],
                            hipMemcpyHostToDevice, stream3));
      H_ERR(hipMemcpyAsync(&ggraph.beg_pos[beg_size_list[block_id3]],
                            &ginst->beg_pos[beg_size_list[block_id3]],
                            beg_size_list[block_id4] - beg_size_list[block_id3],
                            hipMemcpyHostToDevice, stream3));
      check<<<n_blocks, n_threads, 2, stream3>>>(
          Graph_block_size, 2, block_id3, ggraph.adj_list, ggraph.beg_pos,
          ggraph.weight_list, ggraph.vert_count, d_state, d_node_list,
          d_edge_list, d_neigh_l, d_degree_l, n_blocks, d_seed, n_threads,
          d_total, hashtable, bitmap, n_subgraph, node, queue, sample_id,
          depth_tracker, qstart_global, qstop_global, g_sub_index,
	  n_child, depth, each_subgraph, queue_size);
    }

    if (block_active[3]) {
      H_ERR(hipMemcpyAsync(&ggraph.adj_list[adj_size_list[block_id4]],
                            &ginst->adj_list[adj_size_list[block_id4]],
                            adj_size_list[4] - adj_size_list[block_id4],
                            hipMemcpyHostToDevice, stream4));
      H_ERR(hipMemcpyAsync(&ggraph.beg_pos[beg_size_list[block_id4]],
                            &ginst->beg_pos[beg_size_list[block_id4]],
                            beg_size_list[4] - beg_size_list[block_id4],
                            hipMemcpyHostToDevice, stream4));
      check<<<n_blocks, n_threads, 3, stream4>>>(
          Graph_block_size, 3, block_id4, ggraph.adj_list, ggraph.beg_pos,
          ggraph.weight_list, ggraph.vert_count, d_state, d_node_list,
          d_edge_list, d_neigh_l, d_degree_l, n_blocks, d_seed, n_threads,
          d_total, hashtable, bitmap, n_subgraph, node, queue, sample_id,
          depth_tracker, qstart_global, qstop_global, g_sub_index,
	  n_child, depth, each_subgraph, queue_size);
    }
    // wait for completion
    // find new top 3 blocks
    int status1 = hipStreamQuery(stream1);
    // cout<<"Status1: "<<status1<<"\n";
    int status2 = hipStreamQuery(stream2);
    // cout<<"Status2: "<<status2<<"\n";
    HRR(hipDeviceSynchronize());
    HRR(hipMemcpy(start_queue, qstart_global, sizeof(int) * Graph_block,
                   hipMemcpyDeviceToHost));
    HRR(hipMemcpy(seeds_counter, qstop_global, sizeof(int) * Graph_block,
                   hipMemcpyDeviceToHost));
    max = 0, value = 0;
    q_count = 0;
    // display(seeds_counter,Graph_block);
    // display(start_queue,Graph_block);
    int max_value = 0;
    for (int j = 0; j < Graph_block; j++) {
      frontiers_count[j] = seeds_counter[j] - start_queue[j];
      if (frontiers_count[j] <= 0) {
        block_active[j] = 0;
        frontiers_count[j] = 0;
      }
      q_count += frontiers_count[j];
      if (max_value < frontiers_count[j]) {
        max = j;
        max_value = frontiers_count[j];
      }
      //printf("Value: %d, j: %d,Q_count: %d, max: %d\n", frontiers_count[j], j,
      //       q_count, max);
    }
    i++;
    block_active[max] = 1;
    //printf("Value of i:%d\n", i);

    if (q_count == 0) {  // printf("Sampling complete;\n");
      sampling_complete = true;
    }
    if (i == 10) {
      // printf("Sampling complete;\n");
      sampling_complete = true;
    }
    // printf("Total: %d,max:%d, value:%d, Value of i:
    // %d\n",q_count,max,value,i);
  }
  HRR(hipDeviceSynchronize());
  double cmp_time = wtime() - time_start;
  HRR(hipMemcpy(total, g_sub_index, sizeof(int) * n_subgraph,
                 hipMemcpyDeviceToHost));
  int counted = sum(n_subgraph, total);
  float rate = (float)(counted / cmp_time) / 1000000;
  // printf("%s,Kernel time:%f, Rate (Million sampled edges):
  // %f\n",argv[1],cmp_time,rate); printf("%s,Samples: %d,
  // time:%f\n",argv[1],n_subgraph,cmp_time);
  printf("<End of edge list>\n");
  args.sampled_edges = counted;
  args.time = cmp_time;
  return args;
}

// void blocks_allocator(int n_blocks,int *Block,  )
